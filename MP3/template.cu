#include "hip/hip_runtime.h"

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  __shared__ float sibTileM[32][32];
  __shared__ float sibTileN[32][32];

  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  float Pvalue = 0;

  for (int q = 0; q < ceil((float)numAColumns/32); ++q) {
    if (q*32+threadIdx.x < numAColumns) {
      sibTileM[threadIdx.y][threadIdx.x] = A[Row*numAColumns + q*32 + threadIdx.x];
    } else {
      sibTileM[threadIdx.y][threadIdx.x] = 0;
    }

    if (q*32+threadIdx.y < numAColumns) {
      sibTileN[threadIdx.y][threadIdx.x] = B[(q*32+threadIdx.y)*numBColumns + Col];
    } else {
      sibTileN[threadIdx.y][threadIdx.x] = 0;
    }

    __syncthreads();

    for (int k = 0; k < 32; ++k) {
      Pvalue += sibTileM[threadIdx.y][k] * sibTileN[k][threadIdx.x];
    }
    __syncthreads();
  }

  if ((Row < numCRows) && (Col < numCColumns)) {
    C[Row*numCColumns + Col] = Pvalue;
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;

  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows*numCColumns*sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **) &deviceA, numARows*numAColumns*sizeof(float));
  hipMalloc((void **) &deviceB, numBRows*numBColumns*sizeof(float));
  hipMalloc((void **) &deviceC, numCRows*numCColumns*sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows*numAColumns*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows*numBColumns*sizeof(float), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil((float)numCColumns/32), ceil((float)numCRows/32), 1);
  dim3 DimBlock(32, 32, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<DimGrid,DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows*numCColumns*sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
